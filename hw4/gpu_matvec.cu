// $ nvcc -arch=sm_61 gpu03.cu -o gpu03 -Xcompiler -fopenmp
// flag -Xcompiler passes next flag directly to compiler

#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

void vec_mult(double* c, const double* a, const double* b, long M, long N){
  //#pragma omp parallel for schedule(static)
  for (long i = 0; i < M; i++) {
    double sum=0.0;
    for(long j=0; j<N; j++){
     sum += a[i+j*M] * b[j];
    }
    c[i]=sum;
  }
}

__global__
void vec_mult_kernel(double* c, const double* a, const double* b, long M, long N){
  double sum=0.0;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < M){ 
   for(long i=0; i<N; i++) sum += a[idx+i*M] * b[i];
  }
  c[idx]=sum;
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

int main() {
  long N = 10240; // 2^25

  double* x = (double*) malloc(N*N * sizeof(double));
  double* y = (double*) malloc(N * sizeof(double));
  double* z = (double*) malloc(N * sizeof(double));
  double* z_ref = (double*) malloc(N * sizeof(double));
  //#pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    for(long j=0; j<N; j++){
     x[i+j*N] = i+2;
    }
    y[i] = 1.0/(i+1);
    z[i] = 0;
    z_ref[i] = 0;
  }
  double tt = omp_get_wtime();
  vec_mult(z_ref, x, y, N, N);
  printf("CPU %f s\n", omp_get_wtime()-tt);

  double *x_d, *y_d, *z_d;
  hipMalloc(&x_d, N*N*sizeof(double));
  Check_CUDA_Error("malloc x failed");
  hipMalloc(&y_d, N*sizeof(double));
  hipMalloc(&z_d, N*sizeof(double));

  tt = omp_get_wtime();
  hipMemcpy(x_d, x, N*N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(y_d, y, N*sizeof(double), hipMemcpyHostToDevice);
  double ttinner = omp_get_wtime();
  vec_mult_kernel<<<N/1024,1024>>>(z_d, x_d, y_d, N, N);
  hipDeviceSynchronize();
  ttinner = omp_get_wtime() - ttinner;
  hipMemcpy(z, z_d, N*sizeof(double), hipMemcpyDeviceToHost);
  printf("GPU %f s, %f s\n", omp_get_wtime()-tt, ttinner);
  printf("GPU bandwidth is %f GB/s\n", 3*N*N*sizeof(double)/(omp_get_wtime()-tt)/1e9);

  double err = 0;
  for (long i = 0; i < N; i++) err += fabs(z[i]-z_ref[i]);
  printf("Error = %f\n", err);

  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);

  free(x);
  free(y);
  free(z);
  free(z_ref);

  return 0;
}

